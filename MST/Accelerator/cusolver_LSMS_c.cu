#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <complex.h>
//#include "cudaDoubleComplex.hpp"
//#include "DeviceStorage.hpp"

// CUDA error-checking macro for reliable memory allocation and transfers
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n", file, line, 
                        static_cast<unsigned int>(result), hipGetErrorName(result), func);
        exit(EXIT_FAILURE);
    }
}

// CUDA kernel to create a unit matrix
__global__ void createUnitMatrixKernel(hipDoubleComplex *d_matrix, int n) {
    // Calculate the unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check to ensure the thread does not go out of bounds
    if (idx < n * n) {
        // Calculate the row and column from the linear index
        int row = idx / n;
        int col = idx % n;

        // If the thread is on the main diagonal, set the value to 1.0 + 0.0i
        if (row == col) {
            d_matrix[idx] = make_hipDoubleComplex(1.0, 0.0);
        }
        // Otherwise, set the value to 0.0 + 0.0i
        else {
            d_matrix[idx] = make_hipDoubleComplex(0.0, 0.0);
        }
    }
}

bool initialized = false;
hipDoubleComplex  *aDev;
hipDoubleComplex  *aInvDev;
hipDoubleComplex  *tau00Dev;
int bssq = 0;

// CUDA kernel to copy the upper-left nB x nB block of matrix A of size nA x nA into matrix B
__global__ void copyBlockKernel(hipDoubleComplex *A, int nA, hipDoubleComplex *B, int nB) {
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   int col = blockIdx.x * blockDim.x + threadIdx.x;

   // Check if the current thread is within the n x n block
   if (row < nB && col < nB) {
      B[row * nB + col] = A[row * nA + col];
   }
}

extern "C"
void cusolver_lsms_c_(int *m, double _Complex *a, int *block_size, double _Complex *b) {
   // Added on 9/16/2025 ===
   // ======================
   static hipError_t error;
   static int *pivotArray;
   static int *infoArray;
   static int Lwork = 0;
   hipDoubleComplex  *workArray;
   // float time_copyin=0;
   // float time_copyout=0;
   // float time_compute=0;
   hipsolverStatus_t cusolverStatus;
   hipsolverHandle_t cusolverHandle;

   if (!initialized) {
      printf("CUDA memory assigned \n");
      error=hipMalloc((void**)&aDev,  sizeof(hipDoubleComplex)* *m * *m);
      if (error != hipSuccess) fprintf(stderr,"\nError1: %s\n",hipGetErrorString(error));

      error=hipMalloc((void**)&pivotArray,  sizeof(int) * *m);
      if (error != hipSuccess) fprintf(stderr,"\nError2: %s\n",hipGetErrorString(error));

      error=hipMalloc((void**)&infoArray,  sizeof(int));
      if (error != hipSuccess) fprintf(stderr,"\nError3: %s\n",hipGetErrorString(error));

      error=hipMalloc(&aInvDev, sizeof(hipDoubleComplex)* *m * *m);
      if (error != hipSuccess) fprintf(stderr,"\nError4: %s\n",hipGetErrorString(error));

      // Added on 9/16/2025 ===
      bssq = *block_size * *block_size;
      error=hipMalloc(&tau00Dev, sizeof(hipDoubleComplex)*bssq);
      if (error != hipSuccess) fprintf(stderr,"\nError5: %s\n",hipGetErrorString(error));
      // ======================

      initialized = true;
   }
   else if (bssq != *block_size * *block_size) {
      fprintf(stderr,"\nError6: bssq <> block_size**2, %d,%d\n",bssq,*block_size * *block_size);
      exit(1);
   }

   //cudaEvent_t start, stop;
   //cudaEventCreate(&start);
   //cudaEventCreate(&stop);
   hipsolverDnCreate(&cusolverHandle);
   // printf("Lwork is %d\n",Lwork);
   hipsolverDnZgetrf_bufferSize(cusolverHandle, *m, *m, aDev, *m, &Lwork);
   error=hipMalloc((void**)&workArray, Lwork*sizeof(hipDoubleComplex));
   if (error != hipSuccess) fprintf(stderr,"\nError5: %s\n",hipGetErrorString(error));

   //cudaEventRecord(start); 
   error = hipMemcpy(aDev, a, sizeof(hipDoubleComplex)* *m * *m, hipMemcpyHostToDevice);
   if (error != hipSuccess) fprintf(stderr,"\nError6: %s\n",hipGetErrorString(error));
   //cudaEventRecord(stop);
   //cudaEventSynchronize(stop);
   //cudaEventElapsedTime(&time_copyin, start, stop);

   // We are create a unit matrix on device, instead of copying it from the host
   // ========================================
   //* error = cudaMemcpy(aInvDev, tau00, sizeof(cuDoubleComplex)* *m * *m, cudaMemcpyHostToDevice);
   //* if (error != cudaSuccess) fprintf(stderr,"\nError7: %s\n",cudaGetErrorString(error));
   // ----------------------------------------
   // Define kernel launch parameters
   int threads_per_block = 512;
   int num_blocks = (*m * *m + threads_per_block - 1) / threads_per_block;
   createUnitMatrixKernel<<<num_blocks, threads_per_block>>>(aInvDev, *m);
   checkCudaErrors(hipPeekAtLastError());
   // ========================================

   //cudaEventRecord(start);
   cusolverStatus = hipsolverDnZgetrf(cusolverHandle, *m, *m, aDev, *m, workArray, pivotArray, infoArray);
   //if (cusolverStatus == CUSOLVER_STATUS_SUCCESS)
   //  printf("cuSOLVER ZGETRF SUCCESSFUL! \n");
   //else
   //  printf("cuSOLVER ZGETRF UNSUCCESSFUL! \n");

   cusolverStatus = hipsolverDnZgetrs(cusolverHandle,HIPBLAS_OP_N,*m,*m,aDev,*m, pivotArray,aInvDev,*m,infoArray); 
   //if (cusolverStatus == CUSOLVER_STATUS_SUCCESS)
   //  printf("cuSOLVER ZGETRS SUCCESSFUL! \n");
   //else
   //  printf("cuSOLVER ZGETRS UNSUCCESSFUL! \n");
   //cudaEventRecord(stop);
   //cudaEventSynchronize(stop);
   //cudaEventElapsedTime(&time_compute, start, stop);     

   //cudaEventRecord(start);
   // ============================================
   // Comments made on 9/16/2025:
   // The following piece of code is trying to copy each element of the resulting matrix block from device to CPU
   // This is highly inefficient.
   // --------------------------------------------
   // for (int i=0;i<*block_size;i++) {
   //    for (int j=0;j<*block_size;j++) {
   //     cudaMemcpy(&b[i+*block_size * j], &aInvDev[i+*m * j], sizeof(cuDoubleComplex), cudaMemcpyDeviceToHost);
   //    }
   // }
   // --------------------------------------------
   // Instead, we aggregate the data into tau00Dev array and then make one cudaMemcpy call
   // --------------------------------------------
   // Define grid and block dimensions for the kernel
   dim3 threadsPerBlock(32, 32); // Using a 32x32 block
   dim3 blocksPerGrid((*block_size + threadsPerBlock.x - 1) / threadsPerBlock.x,
                      (*block_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

   // Launch the kernel
   copyBlockKernel<<<blocksPerGrid, threadsPerBlock>>>(aInvDev, *m, tau00Dev, *block_size);
   hipMemcpy(b, tau00Dev, sizeof(hipDoubleComplex)*bssq, hipMemcpyDeviceToHost);
   // ============================================
   //cudaEventRecord(stop);
   //cudaEventSynchronize(stop);
   //cudaEventElapsedTime(&time_copyout, start, stop);

   //Print the time (in ms) for GPU data transfer and GPU compute
   //printf("Time for copyin: %f\tfor copyout: %f\tfor compute inverse: %f\n",
   //                       time_copyin*0.001,time_copyout*0.001,time_compute*0.001);

   //clean up
   hipsolverDnDestroy(cusolverHandle);
   hipFree(workArray);
}
